#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#define BLOCK_COUNT 2048
#define MILLER_RABIN_THREAD_COUNT 64

__constant__ uint32_t d_num_base[2];
__device__ uint32_t shifted_bits[BLOCK_COUNT];
__device__ uint32_t v_exp_mod[BLOCK_COUNT];
__global__ void init_num(uint32_t *num_all){
	int idx = blockIdx.x;
	uint32_t num = d_num_base[0] + idx * 2, u = num - 1;
	num_all[idx] = num;
	uint32_t k = 0;
	while(!(u & 1)) {
		k++;
		u >>= 1;
	}
	shifted_bits[blockIdx.x] = k;
}
__device__ __forceinline__ uint32_t mul_mod(uint32_t a0, uint32_t b0, uint32_t n0) {
	double a = a0, b = b0, n = n0;
	double q = __fma_rn(a * b, (double)1.0/n, 6755399441055744.0);
	q -= 6755399441055744.0;
	double h = n * q, l = __fma_rn(n, q, -h);
	double rem = __fma_rn(a, b, -h);
	if (rem < 0.0){
		rem += n;
	}
	return (uint32_t)rem;
}
__global__ void test_mul_mod(uint32_t *result){
	//result[0] = mul_mod(0x7FFFFFFF, 0x7FFFFFF0, 0x80000000);
	result[0] = mul_mod(3, 4, 5);
}
__device__ __forceinline__ uint32_t exp_mod(uint32_t a0, uint32_t b, uint32_t n) {
	uint32_t a = a0, result = 1;
	while (b) {
		if (b & 1) {
			result = mul_mod(result, a, n);
		}
		a = mul_mod(a, a, n);
		b /= 2;
	}
	return result;
}
__device__ __forceinline__ int _miller_rabin_test(uint32_t x, uint32_t shifted_bits, uint32_t n) {
	uint32_t pre = x;
	for (int i = 0; i < shifted_bits; i++) {
		x = mul_mod(x, x, n);
		if ((1 == x) && (pre != 1) && (pre != (n - 1))) {
			return 0;
		} else if (1 == x) {
			break;
		}
		pre = x;
	}
	if (x != 1) {
		return 0;
	}
	return 1;
}
__global__ void miller_rabin_test(uint32_t *num_all, float *d_rand_num) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t n = num_all[blockIdx.x];
	uint32_t x = (uint32_t)(d_rand_num[idx] * (n - 2)) + 2;
	x = exp_mod(x, (n - 1) >> shifted_bits[blockIdx.x], n);
	if (!_miller_rabin_test(x, shifted_bits[blockIdx.x], num_all[blockIdx.x])) {
		num_all[blockIdx.x] = 0;
	}
}

int main(int argc, char *argv[]) {
//	uint32_t *d_mul_mod, mul_mod[1];
//	cudaMalloc(&d_mul_mod, sizeof(uint32_t));
//	test_mul_mod<<<1, 1, 0>>>(d_mul_mod);
//	cudaMemcpy(mul_mod, d_mul_mod, sizeof(uint32_t), cudaMemcpyDeviceToHost);
//	printf("%lu\n", mul_mod[0]);
//	return -1;

	uint32_t start, stop;
	if (argc != 3) {
		fputs("Invalid arguments.\n", stderr);
		return 1;
	} else {
		start = strtoul(argv[1], NULL, 0);
		stop = strtoul(argv[2], NULL, 0);
		if (start < 3) {
			start = 3;
		} else if (!(start & 1)) {
			start += 1;
		}
		if (start >= stop) {
			fputs("Invalid range.\n", stderr);
			return 1;
		}
	}

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
	float *d_rand_num;
	hipMalloc((void **)&d_rand_num, BLOCK_COUNT * MILLER_RABIN_THREAD_COUNT*sizeof(float));

	uint32_t *d_num, num[BLOCK_COUNT];
	hipMalloc(&d_num, sizeof(uint32_t) * BLOCK_COUNT);

	uint32_t base = start, data[2];
	bool running = false;
	while (base < stop) {
		data[0] = base;
		data[1] = stop;
		if (running) {
			hipMemcpy(num, d_num, sizeof(uint32_t) * BLOCK_COUNT, hipMemcpyDeviceToHost);
		}
		hipMemcpyToSymbol(HIP_SYMBOL(d_num_base), data, sizeof(uint32_t) * 2);
		init_num<<<BLOCK_COUNT, 1, 0>>>(d_num);
		hiprandGenerateUniform(gen, d_rand_num, BLOCK_COUNT * MILLER_RABIN_THREAD_COUNT);
		miller_rabin_test<<<BLOCK_COUNT, MILLER_RABIN_THREAD_COUNT, 0>>>(d_num, d_rand_num);
		if (running) {
			for (int i = 0; i < BLOCK_COUNT; i++) {
				if (num[i] > 0 && num[i] < stop) {
					printf("%lu\n", num[i]);
				}
			}
		}
		running = true;
		base += BLOCK_COUNT * 2;
	}
	hipFree(d_num);
	return 0;
}

